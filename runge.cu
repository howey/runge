#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include "runge.h"

/* Time is in units of ns */
static const double ALPHA = 0.02; 
static const double GAMMA = 1.76e-2;
static const double KANIS = 1e6;
static const double TIMESTEP = (1e-5);
static const double MSAT = 500.0;
static const double JEX = 1;
static const double VOL = 2.7e-23;
static const double TEMP = 300.0;
static const double BOLTZ = 1.38e-34;

static double *xx;
static SphVector **y;
static Vector H;
static Vector *H_d;
static SphVector *yout_d;
static hiprandStateXORWOW_t *state;

__global__ void initializeRandom(hiprandStateXORWOW_t * state, int nvar, unsigned long long seed) {
	//the thread id
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	//initialize RNG
	if(i < nvar)
		hiprand_init(seed, i, 0, &state[i]);
}
__global__ void rk4First(SphVector *yt_d, SphVector *y_d, SphVector * dydx_d, double hh, int n) {
	//TODO: Use shared memory
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i < n) {
		yt_d[i].r = y_d[i].r + hh * dydx_d[i].r;
		yt_d[i].phi = y_d[i].phi + hh * dydx_d[i].phi;
		yt_d[i].theta = y_d[i].theta + hh * dydx_d[i].theta;
	}
}

__global__ void rk4Second(SphVector *yt_d, SphVector *y_d, SphVector *dyt_d, double hh, int n) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i < n) {
		yt_d[i].r = y_d[i].r + hh * dyt_d[i].r;
		yt_d[i].phi = y_d[i].phi + hh * dyt_d[i].phi;
		yt_d[i].theta = y_d[i].theta + hh * dyt_d[i].theta;
	}
}

__global__ void rk4Third(SphVector *yt_d, SphVector *y_d, SphVector *dym_d, SphVector * dyt_d, double h, int n) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i < n) {
		yt_d[i].r = y_d[i].r + h * dym_d[i].r;
		dym_d[i].r += dyt_d[i].r;
		yt_d[i].phi = y_d[i].phi + h * dym_d[i].phi;
		dym_d[i].phi += dyt_d[i].phi;
		yt_d[i].theta = y_d[i].theta + h * dym_d[i].theta;
		dym_d[i].theta += dyt_d[i].theta;
	}
}

__global__ void rk4Fourth(SphVector *yout_d, SphVector *y_d, SphVector *dydx_d, SphVector *dyt_d, SphVector *dym_d, double h6, int n) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i < n) {
		yout_d[i].r = y_d[i].r + h6 * (dydx_d[i].r + dyt_d[i].r + 2.0 * dym_d[i].r);
		yout_d[i].phi = y_d[i].phi + h6 * (dydx_d[i].phi + dyt_d[i].phi + 2.0 * dym_d[i].phi);
		yout_d[i].theta = y_d[i].theta + h6 * (dydx_d[i].theta + dyt_d[i].theta + 2.0 * dym_d[i].theta);
	}
}

//Shamelessly copied from Numerical Recipes
/*
Given values for the variables y[1..n] and their derivatives dydx[1..n] known at x , use the
fourth-order Runge-Kutta method to advance the solution over an interval h and return the
incremented variables as yout[1..n] , which need not be a distinct array from y . The user
supplies the routine derivs(x,y,dydx) , which returns derivatives dydx at x .
*/
void rk4(SphVector y_d[], SphVector dydx_d[], int n, double x, double h, SphVector yout[], void (*derivs)(double, SphVector[], SphVector[], int, Vector[])) {
	double xh, hh, h6; 

	//device arrays
	SphVector *dym_d, *dyt_d, *yt_d;

	//allocate device arrays
	hipMalloc((void **)&dym_d, sizeof(SphVector) * n);
	hipMalloc((void **)&dyt_d, sizeof(SphVector) * n);
	hipMalloc((void **)&yt_d, sizeof(SphVector) * n);

	hh = h * 0.5;
	h6 = h / 6.0;
	xh = x + hh;

	//First step
	rk4First<<<ceil(n/512.0), 512>>>(yt_d, y_d, dydx_d, hh, n);

	//Second step
	(*derivs)<<<ceil(n/512.0), 512>>>(xh, yt_d, dyt_d, n, H_d);
	rk4Second<<<ceil(n/512.0), 512>>>(yt_d, y_d, dyt_d, hh, n);

	//Third step
	(*derivs)<<<ceil(n/512.0), 512>>>(xh, yt_d, dym_d, n, H_d);
	rk4Third<<<ceil(n/512.0), 512>>>(yt_d, y_d, dym_d, dyt_d, h, n);

	//Fourth step
	(*derivs)<<<ceil(n/512.0), 512>>>(x + h, yt_d, dyt_d, n, H_d);
	//Accumulate increments with proper weights
	rk4Fourth<<<ceil(n/512.0), 512>>>(yout_d, y_d, dydx_d, dyt_d, dym_d, h6, n);

	//Copy yout to host
	hipMemcpy(yout, yout_d, sizeof(SphVector) * n, hipMemcpyDeviceToHost);
	
	//Free device arrays
	hipFree(yt_d);
	hipFree(dyt_d);
	hipFree(dym_d);
	//hipFree(yout_d);
}

//Computes the local applied field for every atom of moment M. The global applied field is passed in as H. 
__global__ void computeField(Vector * H_d, Vector H, SphVector * M, int nvar, hiprandStateXORWOW_t * state) {
	//Thread coordinates
	int tx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	int ty = blockIdx.y * BLOCK_SIZE + threadIdx.y;
	int tz = blockIdx.z * BLOCK_SIZE + threadIdx.z;
	int i = tz * WIDTH * HEIGHT +  ty * WIDTH + tx;
	
	if(i < nvar) {
		//the applied field
		H_d[i].x = H.x;
		H_d[i].y = H.y;
		H_d[i].z = H.z;

		//the anisotropy field
		H_d[i].x += (1/M[i].r) * -2 * KANIS * cos(M[i].theta) * sin(M[i].theta) * cos(M[i].phi) * cos(M[i].theta);
		H_d[i].y += (1/M[i].r) * -2 * KANIS * cos(M[i].theta) * sin(M[i].theta) * sin(M[i].phi) * cos(M[i].theta);
		H_d[i].z += (1/M[i].r) * 2 * KANIS * cos(M[i].theta) * sin(M[i].theta) * sin(M[i].theta);

		//the field from random thermal motion
		//TODO: sd doesn't hve to be computed each time, it is constant
		#if USE_THERMAL
		double sd = (1e9) * sqrt((2 * BOLTZ * TEMP * ALPHA)/(GAMMA * VOL * MSAT * TIMESTEP)); //time has units of s here
		double thermX = sd * hiprand_normal_double(&state[i]); 
		double thermY = sd * hiprand_normal_double(&state[i]);
		double thermZ = sd * hiprand_normal_double(&state[i]);

		H_d[i].x += thermX;
		H_d[i].y += thermY;
		H_d[i].z += thermZ;
		#endif

		//the exchange field
		SphVector up, down, left, right, front, back;

		if(i % (WIDTH * HEIGHT) < WIDTH)
			up = M[i + WIDTH * (HEIGHT - 1)]; 
		else
			up = M[i - WIDTH];

		if(i % (WIDTH * HEIGHT) > (WIDTH * (HEIGHT - 1) - 1))
			down = M[i - WIDTH * (HEIGHT - 1)];
		else
			down = M[i + WIDTH];	

		if(i % WIDTH == 0)
			left = M[i + (WIDTH - 1)];
		else
			left = M[i - 1];

		if((i + 1) % WIDTH == 0)
			right = M[i - (WIDTH - 1)];
		else
			right = M[i + 1];

		if(i < (WIDTH * HEIGHT))
			front = M[i + (WIDTH * HEIGHT * (DEPTH - 1))];
		else
			front = M[i - (WIDTH * HEIGHT)];

		if(i >= (WIDTH * HEIGHT * (DEPTH - 1)))
			back = M[i - (WIDTH * HEIGHT * (DEPTH - 1))];
		else
			back = M[i + (WIDTH * HEIGHT)];

		H_d[i].x += JEX * (sin(up.theta) * cos(up.phi) + sin(down.theta) * cos(down.phi) + sin(left.theta) * cos(left.phi) + sin(right.theta) * cos(right.phi) + sin(front.theta) * cos(front.phi) + sin(back.theta) * cos(back.phi));
		H_d[i].y += JEX * (sin(up.theta) * sin(up.phi) + sin(down.theta) * sin(down.phi) + sin(left.theta) * sin(left.phi) + sin(right.theta) * sin(right.phi) + sin(front.theta) * sin(front.phi) + sin(back.theta) * sin(back.phi)); 
		H_d[i].z += JEX * (cos(up.phi) + cos(down.phi) + cos(left.phi) + cos(right.phi) + cos(front.phi) + cos(back.phi));
	}
}

__global__ void mDot(double t, SphVector M[], SphVector dMdt[], int nvar, Vector H[]) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	//Compute derivative
	if(i < nvar) {
		dMdt[i].r = 0;
		dMdt[i].phi = GAMMA * ((cos(M[i].theta) * sin(M[i].phi) * H[i].y) / sin(M[i].theta) + (cos(M[i].theta) * cos(M[i].phi) * H[i].x) / sin(M[i].theta) - H[i].z) + ((ALPHA * GAMMA)/(1 + ALPHA * ALPHA)) * ((cos(M[i].phi) * H[i].y) / sin(M[i].theta) - (sin(M[i].phi) * H[i].x) / sin(M[i].theta));
		dMdt[i].theta = -GAMMA * (cos(M[i].phi) * H[i].y - sin(M[i].phi) * H[i].x) + ((ALPHA * GAMMA)/(1 + ALPHA * ALPHA)) * (cos(M[i].theta) * cos(M[i].phi) * H[i].x - H[i].z * sin(M[i].theta) + cos(M[i].theta) * sin(M[i].phi) * H[i].y);
	}
}

/*
Starting from initial values vstart[0..nvar-1] known at x1 use fourth-order Runge-Kutta
to advance nstep equal increments to x2. The user-supplied routine derivs(x,v,dvdx)
evaluates derivatives. Results are stored in the global variables y[0..nvar-1][0..nstep]
and xx[0..nstep].
*/
void rkdumb(SphVector vstart[], int nvar, double x1, double x2, int nstep, void (*derivs)(double, SphVector[], SphVector[], int, Vector[])) {
	double x, h;
	dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
        dim3 gridDim(ceil(WIDTH/BLOCK_SIZE), ceil(HEIGHT/BLOCK_SIZE), ceil(DEPTH/BLOCK_SIZE));	
	SphVector *v, *vout, *dv;

	//device arrays
	SphVector *v_d, *dv_d;

	v = (SphVector *)malloc(sizeof(SphVector) * nvar);
	vout = (SphVector *)malloc(sizeof(SphVector) * nvar);
	dv = (SphVector *)malloc(sizeof(SphVector) * nvar);

	hipMalloc((void **)&yout_d, sizeof(SphVector) * nvar);

	//allocate device memory for mDot
	hipMalloc((void **)&v_d, sizeof(SphVector) * nvar);
	hipMalloc((void **)&dv_d, sizeof(SphVector) * nvar);
	hipMalloc((void **)&H_d, sizeof(SphVector) * nvar);
	
	for (int i = 0;i < nvar;i++) { 
		v[i] = vstart[i];
		y[i][0] = v[i]; 
	}

	xx[0] = x1;
	x = x1;
	h = (x2-x1)/nstep;

	for (int k = 0; k < nstep; k++) {

		//Copy memory to device
		//After the first timestep, the value of v and yout_d are the same. d2d memcpy is much faster than h2s, so do it instead
		if(k == 0) hipMemcpy(v_d, v, sizeof(SphVector) * nvar, hipMemcpyHostToDevice);
		else hipMemcpy(v_d, yout_d, sizeof(SphVector) * nvar, hipMemcpyDeviceToDevice);

		//Launch kernel to compute H field
		computeField<<<gridDim, blockDim>>>(H_d, H, v_d, nvar, state); 

		//Launch kernel to compute derivatives
		(*derivs)<<<ceil(nvar/512.0), 512>>>(x, v_d, dv_d, nvar, H_d);
		
		rk4(v_d,dv_d,nvar,x,h,vout,derivs);
		if ((double)(x + h) == x) fprintf(stderr, "Step size too small in routine rkdumb");
		x += h;
		xx[k + 1] = x;
		for (int i = 0; i < nvar; i++) {
			v[i] = vout[i];
			y[i][k + 1] = v[i];
		}

	}

	free(dv);
	free(vout);
	free(v);
	hipFree(yout_d);
	hipFree(v_d);
	hipFree(dv_d);
	hipFree(H_d);
}

int main(int argc, char *argv[]) {
	int nvar = HEIGHT * WIDTH * DEPTH; //M for each particle 
	int nstep;
	double endTime;
	SphVector vstart[nvar]; 
	FILE * output = fopen("output.txt", "w");
	
	//Initialize random number generator
	unsigned long long seed = time(NULL);
	hipMalloc((void **)&state, sizeof(hiprandStateXORWOW_t) * nvar);
	initializeRandom<<<ceil(nvar/512.0), 512>>>(state, nvar, seed);

	if(output == NULL) {
		printf("error opening file\n");
		return 0;
	}
	
	for(int i = 0; i < nvar; i++) {	
		vstart[i].r = MSAT;
		vstart[i].theta = 0.01;
		vstart[i].phi = 0;
	}

	Vector Happl = {0.0, 0.0, 5000.0};

	//Get the step size for the simulation 
	if(argc < 2) {
		printf("Usage: %s [step size]\n", argv[0]);
		return 1;
	}

	endTime = (1e9)*strtof(argv[1], NULL); //In ns
	
	endTime /= 100; //Reduce memory usage

	nstep = ((int)ceil(endTime/TIMESTEP));

	xx = (double *)malloc(sizeof(double) * (nstep + 1));
	y = (SphVector **)malloc(sizeof(SphVector *) * nvar); 
	for(int i = 0; i < nvar; i++) {
		y[i] = (SphVector *)malloc(sizeof(SphVector) * (nstep + 1));
	}

	
	bool isDecreasing = true;
	for(int i = 0; i <= 400; i++) {
		//Applied field
		H.x = Happl.x;
		H.y = Happl.y;
		H.z = Happl.z;

		for(int j = 0; j < 100; j++) {
			//Simulate!
			rkdumb(vstart, nvar, endTime * j, endTime * (j + 1) - TIMESTEP, nstep, mDot); 

			for(int i = 0; i < nvar; i++) {
				vstart[i].r = y[i][nstep].r;
				vstart[i].theta = y[i][nstep].theta;
				vstart[i].phi = y[i][nstep].phi;
			}
		}
	
		double mag = 0.0;	
		for(int k = 0; k < nvar; k++) {
			mag += (y[k][nstep].r)*cos(y[k][nstep].theta);
			//fprintf(output, "%f\t%f\n", Happl.z, (y[k][nstep].r)*cos(y[k][nstep].theta));
		}
		mag /= (double)nvar;
		fprintf(output, "%f\t%f\n", Happl.z, mag);

		//Adjust applied field strength at endTime intervals	
		if(Happl.z + 5000.0 < 1.0) isDecreasing = false;
		isDecreasing ? (Happl.z -= 50.0) : (Happl.z += 50.0);
	}
	//Probably don't really need these since we're about to exit the program
	free(xx);
	free(y);
	hipFree(state);
	return 0;
}
