#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include "runge.h"

//CUDA call error checking
//From https://stackoverflow.com/questions/14038589
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char * file, int line, bool abort=true) {
	if(code != hipSuccess) {
		fprintf(stderr, "GPU Assert!: %s File: %s Line: %d\n", hipGetErrorString(code), file, line);
		if(abort)
			exit(code);
	}
}

/* Time is in units of ns */
static const double ALPHA = 0.02; //dimensionless
static const double GAMMA = 1.76e-2; //(Oe*ns)^-1
static const double KANIS = 7.0e7; //erg*cm^-3
static const double TIMESTEP = (1e-7); //ns, the integrator timestep
static const double MSAT = 1100.0; //emu*cm^-3
static const double JEX = 1.1e-6; //erg*cm^-1
static const double ALEN = 3e-8; //cm
static const double TEMP = 0.0; //K
static const double BOLTZ = 1.38e-34; //g*cm^2*ns^-2*K^-1
static const double FIELDSTEP = 500.0; //Oe, the change in the applied field
static const double FIELDTIMESTEP = 0.1; //ns, time to wait before changing applied field
static const double FIELDRANGE = 130000.0; //Oe, create loop from FIELDRANGE to -FIELDRANGE Oe

static double *xx;
static SphVector **y;
static Vector H;
static Vector *H_d;
static SphVector *v_d;
static SphVector *yout_d;
static hiprandStateXORWOW_t *state;

__global__ void initializeRandom(hiprandStateXORWOW_t * state, int nvar, unsigned long long seed) {
	//the thread id
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	//initialize RNG
	if(i < nvar)
		hiprand_init(seed, i, 0, &state[i]);
}

__global__ void rk4Kernel(SphVector * y_d, int n, double x, double h, SphVector * yout_d, Vector * H) {
	//intermediate rk4 steps
	__shared__ SphVector dym_d;
	__shared__ SphVector dyt_d;
	__shared__ SphVector yt_d;
	__shared__ Vector H_s;
	__shared__ SphVector y_s;
	__shared__ SphVector dydx_s;

	double hh, h6;
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	
	//To avoid round-off errors, scale field and time
	double scale = (2.0 * KANIS / MSAT);
	h *= scale;
	
	hh = h * 0.5;
	h6 = h / 6.0;

	//Load field 
	if(i < n) {
		H_s = H[i];
		H_s.x /= scale;
		H_s.y /= scale;
		H_s.z /= scale;
	}

	//Load input
	if(i < n) {
		y_s = y_d[i];
	}

	//First step
	dydx_s.r = 0;
	dydx_s.phi = GAMMA * ((cos(y_s.theta) * sin(y_s.phi) * H_s.y) / sin(y_s.theta) + (cos(y_s.theta) * cos(y_s.phi) * H_s.x) / sin(y_s.theta) - H_s.z) + ((ALPHA * GAMMA)/(1 + ALPHA * ALPHA)) * ((cos(y_s.phi) * H_s.y) / sin(y_s.theta) - (sin(y_s.phi) * H_s.x) / sin(y_s.theta));
	dydx_s.theta = -GAMMA * (cos(y_s.phi) * H_s.y - sin(y_s.phi) * H_s.x) + ((ALPHA * GAMMA)/(1 + ALPHA * ALPHA)) * (cos(y_s.theta) * cos(y_s.phi) * H_s.x - H_s.z * sin(y_s.theta) + cos(y_s.theta) * sin(y_s.phi) * H_s.y);

	yt_d.r = y_s.r + hh * dydx_s.r;
	yt_d.phi = y_s.phi + hh * dydx_s.phi;
	yt_d.theta = y_s.theta + hh * dydx_s.theta;

	//Second step
	dyt_d.r = 0;
	dyt_d.phi = GAMMA * ((cos(yt_d.theta) * sin(yt_d.phi) * H_s.y) / sin(yt_d.theta) + (cos(yt_d.theta) * cos(yt_d.phi) * H_s.x) / sin(yt_d.theta) - H_s.z) + ((ALPHA * GAMMA)/(1 + ALPHA * ALPHA)) * ((cos(yt_d.phi) * H_s.y) / sin(yt_d.theta) - (sin(yt_d.phi) * H_s.x) / sin(yt_d.theta));
	dyt_d.theta = -GAMMA * (cos(yt_d.phi) * H_s.y - sin(yt_d.phi) * H_s.x) + ((ALPHA * GAMMA)/(1 + ALPHA * ALPHA)) * (cos(yt_d.theta) * cos(yt_d.phi) * H_s.x - H_s.z * sin(yt_d.theta) + cos(yt_d.theta) * sin(yt_d.phi) * H_s.y);

	yt_d.r = y_s.r + hh * dyt_d.r;
	yt_d.phi = y_s.phi + hh * dyt_d.phi;
	yt_d.theta = y_s.theta + hh * dyt_d.theta;

	//Third step
	dym_d.r = 0;
	dym_d.phi = GAMMA * ((cos(yt_d.theta) * sin(yt_d.phi) * H_s.y) / sin(yt_d.theta) + (cos(yt_d.theta) * cos(yt_d.phi) * H_s.x) / sin(yt_d.theta) - H_s.z) + ((ALPHA * GAMMA)/(1 + ALPHA * ALPHA)) * ((cos(yt_d.phi) * H_s.y) / sin(yt_d.theta) - (sin(yt_d.phi) * H_s.x) / sin(yt_d.theta));
	dym_d.theta = -GAMMA * (cos(yt_d.phi) * H_s.y - sin(yt_d.phi) * H_s.x) + ((ALPHA * GAMMA)/(1 + ALPHA * ALPHA)) * (cos(yt_d.theta) * cos(yt_d.phi) * H_s.x - H_s.z * sin(yt_d.theta) + cos(yt_d.theta) * sin(yt_d.phi) * H_s.y);

	yt_d.r = y_s.r + h * dym_d.r;
	dym_d.r += dyt_d.r;
	yt_d.phi = y_s.phi + h * dym_d.phi;
	dym_d.phi += dyt_d.phi;
	yt_d.theta = y_s.theta + h * dym_d.theta;
	dym_d.theta += dyt_d.theta;

	//Fourth step
	dyt_d.r = 0;
	dyt_d.phi = GAMMA * ((cos(yt_d.theta) * sin(yt_d.phi) * H_s.y) / sin(yt_d.theta) + (cos(yt_d.theta) * cos(yt_d.phi) * H_s.x) / sin(yt_d.theta) - H_s.z) + ((ALPHA * GAMMA)/(1 + ALPHA * ALPHA)) * ((cos(yt_d.phi) * H_s.y) / sin(yt_d.theta) - (sin(yt_d.phi) * H_s.x) / sin(yt_d.theta));
	dyt_d.theta = -GAMMA * (cos(yt_d.phi) * H_s.y - sin(yt_d.phi) * H_s.x) + ((ALPHA * GAMMA)/(1 + ALPHA * ALPHA)) * (cos(yt_d.theta) * cos(yt_d.phi) * H_s.x - H_s.z * sin(yt_d.theta) + cos(yt_d.theta) * sin(yt_d.phi) * H_s.y);

	if(i < n) {
		yout_d[i].r = y_s.r + h6 * (dydx_s.r + dyt_d.r + 2.0 * dym_d.r);
		yout_d[i].phi = y_s.phi + h6 * (dydx_s.phi + dyt_d.phi + 2.0 * dym_d.phi);
		yout_d[i].theta = y_s.theta + h6 * (dydx_s.theta + dyt_d.theta + 2.0 * dym_d.theta);
	}
}

//Computes the local applied field for every atom of moment M. The global applied field is passed in as H. 
__global__ void computeField(Vector * H_d, Vector H, SphVector * M, int nvar, hiprandStateXORWOW_t * state) {
	/* Declare shared memory for CUDA block.
	   Since a halo element neighbors only one atom,
	   halo elements are not loaded into shared memory.
	   Instead, they are read from global memory as usual. */
	__shared__ SphVector M_s[BLOCK_SIZE][BLOCK_SIZE][BLOCK_SIZE];

	//Thread coordinates
	int tx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	int ty = blockIdx.y * BLOCK_SIZE + threadIdx.y;
	int tz = blockIdx.z * BLOCK_SIZE + threadIdx.z;
	int i = tz * WIDTH * HEIGHT +  ty * WIDTH + tx;
	Vector H_t;

	if(tx < WIDTH && ty < HEIGHT && tz < DEPTH) {
		//Load block into shared memory
		M_s[threadIdx.z][threadIdx.y][threadIdx.x] = M[i];
		__syncthreads();

		//the applied field
		H_t.x = H.x;
		H_t.y = H.y;
		H_t.z = H.z;

		//the anisotropy field
		H_t.x += (1/M_s[threadIdx.z][threadIdx.y][threadIdx.x].r) * -2 * KANIS * cos(M_s[threadIdx.z][threadIdx.y][threadIdx.x].theta) * sin(M_s[threadIdx.z][threadIdx.y][threadIdx.x].theta) * cos(M_s[threadIdx.z][threadIdx.y][threadIdx.x].phi) * cos(M_s[threadIdx.z][threadIdx.y][threadIdx.x].theta);
		H_t.y += (1/M_s[threadIdx.z][threadIdx.y][threadIdx.x].r) * -2 * KANIS * cos(M_s[threadIdx.z][threadIdx.y][threadIdx.x].theta) * sin(M_s[threadIdx.z][threadIdx.y][threadIdx.x].theta) * sin(M_s[threadIdx.z][threadIdx.y][threadIdx.x].phi) * cos(M_s[threadIdx.z][threadIdx.y][threadIdx.x].theta);
		H_t.z += (1/M_s[threadIdx.z][threadIdx.y][threadIdx.x].r) * 2 * KANIS * cos(M_s[threadIdx.z][threadIdx.y][threadIdx.x].theta) * sin(M_s[threadIdx.z][threadIdx.y][threadIdx.x].theta) * sin(M_s[threadIdx.z][threadIdx.y][threadIdx.x].theta);

		//the field from random thermal motion
		double vol = ALEN * ALEN * ALEN;
		double sd = (1e9) * sqrt((2 * BOLTZ * TEMP * ALPHA)/(GAMMA * vol * MSAT * TIMESTEP)); //time has units of s here

		double thermX = sd * hiprand_normal_double(&state[i]); 
		double thermY = sd * hiprand_normal_double(&state[i]);
		double thermZ = sd * hiprand_normal_double(&state[i]);

		H_t.x += thermX;
		H_t.y += thermY;
		H_t.z += thermZ;


		//the exchange field
		SphVector up, down, left, right, front, back;

		//if(i % (WIDTH * HEIGHT) < WIDTH) //if at top of particle
		if(ty == 0)
			up = M[i + WIDTH * (HEIGHT - 1)]; 
		else if(threadIdx.y > 0)
			up = M_s[threadIdx.z][threadIdx.y - 1][threadIdx.x];
		else
			up = M[i - WIDTH];

		//if(i % (WIDTH * HEIGHT) > (WIDTH * (HEIGHT - 1) - 1)) //if at bottom of particle
		if(ty == (HEIGHT - 1))
			down = M[i - WIDTH * (HEIGHT - 1)];
		else if(threadIdx.y < (blockDim.y - 1))
			down = M_s[threadIdx.z][threadIdx.y + 1][threadIdx.x];
		else
			down = M[i + WIDTH];	

		//if(i % WIDTH == 0) //if at left
		if(tx == 0)
			left = M[i + (WIDTH - 1)]; 
		else if(threadIdx.x > 0)
			left = M_s[threadIdx.z][threadIdx.y][threadIdx.x - 1];
		else
			left = M[i - 1];

		//if((i + 1) % WIDTH == 0) //if at right
		if(tx == (WIDTH - 1))
			right = M[i - (WIDTH - 1)];
		else if(threadIdx.x < (blockDim.x - 1))
			right = M_s[threadIdx.z][threadIdx.y][threadIdx.x + 1];
		else
			right = M[i + 1];

		//if(i < (WIDTH * HEIGHT)) //if at front
		if(tz == 0)
			front = M[i + (WIDTH * HEIGHT * (DEPTH - 1))];
		else if(threadIdx.z > 0)
			front = M_s[threadIdx.z - 1][threadIdx.y][threadIdx.x];
		else
			front = M[i - (WIDTH * HEIGHT)];

		//if(i >= (WIDTH * HEIGHT * (DEPTH - 1))) //if at rear
		if(tz == (DEPTH - 1))
			back = M[i - (WIDTH * HEIGHT * (DEPTH - 1))];
		else if(threadIdx.z < (blockDim.z - 1))
			back = M_s[threadIdx.z + 1][threadIdx.y][threadIdx.x];
		else
			back = M[i + (WIDTH * HEIGHT)];

		double Hex = 2.0 * JEX / (MSAT * ALEN * ALEN);

		H_t.x += Hex * (sin(up.theta) * cos(up.phi) + sin(down.theta) * cos(down.phi) + sin(left.theta) * cos(left.phi) + sin(right.theta) * cos(right.phi) + sin(front.theta) * cos(front.phi) + sin(back.theta) * cos(back.phi));
		H_t.y += Hex * (sin(up.theta) * sin(up.phi) + sin(down.theta) * sin(down.phi) + sin(left.theta) * sin(left.phi) + sin(right.theta) * sin(right.phi) + sin(front.theta) * sin(front.phi) + sin(back.theta) * sin(back.phi)); 
		H_t.z += Hex * (cos(up.theta) + cos(down.theta) + cos(left.theta) + cos(right.theta) + cos(front.theta) + cos(back.theta));

		__syncthreads();
		H_d[i] = H_t;
	}
}

/*
Starting from initial values vstart[0..nvar-1] known at x1 use fourth-order Runge-Kutta
to advance nstep equal increments to x2. The user-supplied routine derivs(x,v,dvdx)
evaluates derivatives. Results are stored in the global variables y[0..nvar-1][0..nstep]
and xx[0..nstep].
*/
void rkdumb(SphVector vstart[], int nvar, double x1, double x2, int nstep) {
	double x, h;
	dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
        dim3 gridDim(ceil(((float)WIDTH)/((float)BLOCK_SIZE)), ceil(((float)HEIGHT)/((float)BLOCK_SIZE)), ceil(((float)DEPTH)/((float)BLOCK_SIZE)));	
	SphVector *v, *vout;

	v = (SphVector *)malloc(sizeof(SphVector) * nvar);
	vout = (SphVector *)malloc(sizeof(SphVector) * nvar);

	for (int i = 0;i < nvar;i++) { 
		v[i] = vstart[i];
		y[i][0] = v[i]; 
	}

	xx[0] = x1;
	x = x1;
	h = (x2-x1)/nstep;

	for (int k = 0; k < nstep; k++) {

		//Copy memory to device
		//After the first timestep, the value of v and yout_d are the same. d2d memcpy is much faster than h2s, so do it instead
		if(k == 0) {
			#if DEBUG
			gpuErrchk( hipMemcpy(v_d, v, sizeof(SphVector) * nvar, hipMemcpyHostToDevice) );
			#else
			hipMemcpy(v_d, v, sizeof(SphVector) * nvar, hipMemcpyHostToDevice);
			#endif
		}
		else {
			SphVector *t_d = v_d;
			v_d = yout_d;
			yout_d = t_d;
		}

		//Launch kernel to compute H field
		computeField<<<gridDim, blockDim>>>(H_d, H, v_d, nvar, state); 

		#if DEBUG
		//Check for kernel launch errors 
		gpuErrchk( hipPeekAtLastError() );
		gpuErrchk( hipDeviceSynchronize() );
		#endif

		rk4Kernel<<<ceil(nvar/VECTOR_SIZE), VECTOR_SIZE>>>(v_d, nvar, x, h, yout_d, H_d);
		
		#if DEBUG
		gpuErrchk( hipPeekAtLastError() );
		gpuErrchk( hipDeviceSynchronize() );
		#endif

		if(k == (nstep - 1)) {
			#if DEBUG
			gpuErrchk( hipMemcpy(vout, yout_d, sizeof(SphVector) * nvar, hipMemcpyDeviceToHost) );
			#else
			hipMemcpy(vout, yout_d, sizeof(SphVector) * nvar, hipMemcpyDeviceToHost);
			#endif
		}

		if ((double)(x + h) == x) 
			fprintf(stderr, "Step size too small in routine rkdumb");

		x += h;
		xx[k + 1] = x;
		for (int i = 0; i < nvar; i++) {
			v[i] = vout[i];
			y[i][k + 1] = v[i];
		}

	}

	free(vout);
	free(v);
}

int main(int argc, char *argv[]) {
	int nvar = HEIGHT * WIDTH * DEPTH; //M for each particle 
	int nstep;
	double endTime;
	SphVector vstart[nvar]; 

	FILE * output = fopen("output.txt", "w");
	if(output == NULL) {
		printf("error opening file\n");
		return 0;
	}

	#if BENCHMARK
	FILE * times = fopen("times.txt", "w");
	if(times == NULL) {
		printf("error opening file: times.txt\n");
		return 1;
	}
	fprintf(times, "Time to simulate %fns\n", FIELDTIMESTEP);
	#endif

	//Initialize random number generator
	unsigned long long seed = time(NULL);
	#if DEBUG
	gpuErrchk( hipMalloc((void **)&state, sizeof(hiprandStateXORWOW_t) * nvar) );
	initializeRandom<<<ceil(nvar/512.0), 512>>>(state, nvar, seed);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
	#else
	hipMalloc((void **)&state, sizeof(hiprandStateXORWOW_t) * nvar);
	initializeRandom<<<ceil(nvar/512.0), 512>>>(state, nvar, seed);
	#endif
	//Configure shared/L1 as 48KB/16KB
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	
	for(int i = 0; i < nvar; i++) {	
		vstart[i].r = MSAT;
		vstart[i].theta = 0.01;
		vstart[i].phi = 0;
	}

	Vector Happl = {0.0, 0.0, FIELDRANGE};
	endTime = FIELDTIMESTEP; 
	endTime /= 100; //Reduce host memory usage
	nstep = ((int)ceil(endTime/TIMESTEP));

	xx = (double *)malloc(sizeof(double) * (nstep + 1));
	y = (SphVector **)malloc(sizeof(SphVector *) * nvar); 
	for(int i = 0; i < nvar; i++) {
		y[i] = (SphVector *)malloc(sizeof(SphVector) * (nstep + 1));
	}

	//Initialize device arrays
	#if DEBUG
	gpuErrchk( hipMalloc((void **)&yout_d, sizeof(SphVector) * nvar) );
	gpuErrchk( hipMalloc((void **)&v_d, sizeof(SphVector) * nvar) );
	gpuErrchk( hipMalloc((void **)&H_d, sizeof(Vector) * nvar) );
	#else
	hipMalloc((void **)&yout_d, sizeof(SphVector) * nvar);
	hipMalloc((void **)&v_d, sizeof(SphVector) * nvar);
	hipMalloc((void **)&H_d, sizeof(Vector) * nvar);
	#endif
	
	bool isDecreasing = true;
	for(int i = 0; i <= (4 * (int)(FIELDRANGE/FIELDSTEP)); i++) {
		//Applied field
		H.x = Happl.x;
		H.y = Happl.y;
		H.z = Happl.z;

		#if BENCHMARK
		time_t start = time(NULL);
		#endif

		for(int j = 0; j < 100; j++) {
			//Simulate!
			rkdumb(vstart, nvar, endTime * j, endTime * (j + 1) - TIMESTEP, nstep); 

			for(int i = 0; i < nvar; i++) {
				vstart[i].r = y[i][nstep].r;
				vstart[i].theta = y[i][nstep].theta;
				vstart[i].phi = y[i][nstep].phi;
			}
		}
	
		#if BENCHMARK
		time_t end = time(NULL);
		fprintf(times, "%lds\n", (long)(end - start));
		fflush(times);
		#endif

		double mag = 0.0;	
		for(int k = 0; k < nvar; k++) {
			mag += (y[k][nstep].r)*cos(y[k][nstep].theta);
		}

		mag /= (double)nvar;
		fprintf(output, "%f\t%f\n", Happl.z, mag);
		fflush(output);

		//Adjust applied field strength at endTime intervals	
		if(Happl.z + FIELDRANGE < 1.0) isDecreasing = false;
		isDecreasing ? (Happl.z -= FIELDSTEP) : (Happl.z += FIELDSTEP);
	}
	//Probably don't really need these since we're about to exit the program
	free(xx);
	free(y);
	#if DEBUG
	gpuErrchk( hipFree(yout_d) );
	gpuErrchk( hipFree(v_d) );
	gpuErrchk( hipFree(H_d) );
	gpuErrchk( hipFree(state) );
	#else
	hipFree(yout_d);
	hipFree(v_d);
	hipFree(H_d);
	hipFree(state);
	#endif
	
	return 0;
}
