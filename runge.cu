#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include "runge.h"

/* Time is in units of ns */
static const double ALPHA = 0.02; //dimensionless
static const double GAMMA = 1.76e-2; //(Oe*ns)^-1
static const double KANIS = 7.0e7; //erg*cm^-3
static const double TIMESTEP = (1e-7); //ns, the integrator timestep
static const double MSAT = 1100.0; //emu*cm^-3
static const double JEX = 1.1e-6; //erg*cm^-1
static const double ALEN = 3e-8; //cm
static const double TEMP = 300.0; //K
static const double BOLTZ = 1.38e-34; //g*cm^2*ns^-2*K^-1
static const double FIELDSTEP = 500.0; //Oe, the change in the applied field
static const double FIELDTIMESTEP = 0.1; //ns, time to wait before changing applied field
static const double FIELDRANGE = 130000.0; //Oe, create loop from FIELDRANGE to -FIELDRANGE Oe

static double *xx;
static SphVector **y;
static Vector H;
static Vector *H_d;
static Vector * Htherm_d;
static SphVector *yout_d;
static hiprandStateXORWOW_t *state;

__global__ void initializeRandom(hiprandStateXORWOW_t * state, int nvar, unsigned long long seed) {
	//the thread id
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	//initialize RNG
	if(i < nvar)
		hiprand_init(seed, i, 0, &state[i]);
}
__global__ void rk4First(SphVector *yt_d, SphVector *y_d, SphVector * dydx_d, double hh, int n) {
	//TODO: Use shared memory
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i < n) {
		yt_d[i].r = y_d[i].r + hh * dydx_d[i].r;
		yt_d[i].phi = y_d[i].phi + hh * dydx_d[i].phi;
		yt_d[i].theta = y_d[i].theta + hh * dydx_d[i].theta;
	}
}

__global__ void rk4Second(SphVector *yt_d, SphVector *y_d, SphVector *dyt_d, double hh, int n) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i < n) {
		yt_d[i].r = y_d[i].r + hh * dyt_d[i].r;
		yt_d[i].phi = y_d[i].phi + hh * dyt_d[i].phi;
		yt_d[i].theta = y_d[i].theta + hh * dyt_d[i].theta;
	}
}

__global__ void rk4Third(SphVector *yt_d, SphVector *y_d, SphVector *dym_d, SphVector * dyt_d, double h, int n) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i < n) {
		yt_d[i].r = y_d[i].r + h * dym_d[i].r;
		dym_d[i].r += dyt_d[i].r;
		yt_d[i].phi = y_d[i].phi + h * dym_d[i].phi;
		dym_d[i].phi += dyt_d[i].phi;
		yt_d[i].theta = y_d[i].theta + h * dym_d[i].theta;
		dym_d[i].theta += dyt_d[i].theta;
	}
}

__global__ void rk4Fourth(SphVector *yout_d, SphVector *y_d, SphVector *dydx_d, SphVector *dyt_d, SphVector *dym_d, double h6, int n) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if(i < n) {
		yout_d[i].r = y_d[i].r + h6 * (dydx_d[i].r + dyt_d[i].r + 2.0 * dym_d[i].r);
		yout_d[i].phi = y_d[i].phi + h6 * (dydx_d[i].phi + dyt_d[i].phi + 2.0 * dym_d[i].phi);
		yout_d[i].theta = y_d[i].theta + h6 * (dydx_d[i].theta + dyt_d[i].theta + 2.0 * dym_d[i].theta);
	}
}

//Computes the local applied field for every atom of moment M. The global applied field is passed in as H, and the thermal motion as Htherm. 
__global__ void computeField(Vector * H_d, Vector H, Vector * Htherm_d, SphVector * M, int nvar) {
	/* Declare shared memory for CUDA block.
	   Since a halo element neighbors only one atom,
	   halo elements are not loaded into shared memory.
	   Instead, they are read from global memory as usual. */
	__shared__ SphVector M_s[BLOCK_SIZE][BLOCK_SIZE][BLOCK_SIZE];

	//Thread coordinates
	int tx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	int ty = blockIdx.y * BLOCK_SIZE + threadIdx.y;
	int tz = blockIdx.z * BLOCK_SIZE + threadIdx.z;
	int i = tz * WIDTH * HEIGHT +  ty * WIDTH + tx;
	Vector H_t;

	if(tx < WIDTH && ty < HEIGHT && tz < DEPTH) {
		//Load block into shared memory
		M_s[threadIdx.z][threadIdx.y][threadIdx.x] = M[i];
		__syncthreads();

		//the applied field
		H_t.x = H.x;
		H_t.y = H.y;
		H_t.z = H.z;

		//the anisotropy field
		H_t.x += (1/M_s[threadIdx.z][threadIdx.y][threadIdx.x].r) * -2 * KANIS * cos(M_s[threadIdx.z][threadIdx.y][threadIdx.x].theta) * sin(M_s[threadIdx.z][threadIdx.y][threadIdx.x].theta) * cos(M_s[threadIdx.z][threadIdx.y][threadIdx.x].phi) * cos(M_s[threadIdx.z][threadIdx.y][threadIdx.x].theta);
		H_t.y += (1/M_s[threadIdx.z][threadIdx.y][threadIdx.x].r) * -2 * KANIS * cos(M_s[threadIdx.z][threadIdx.y][threadIdx.x].theta) * sin(M_s[threadIdx.z][threadIdx.y][threadIdx.x].theta) * sin(M_s[threadIdx.z][threadIdx.y][threadIdx.x].phi) * cos(M_s[threadIdx.z][threadIdx.y][threadIdx.x].theta);
		H_t.z += (1/M_s[threadIdx.z][threadIdx.y][threadIdx.x].r) * 2 * KANIS * cos(M_s[threadIdx.z][threadIdx.y][threadIdx.x].theta) * sin(M_s[threadIdx.z][threadIdx.y][threadIdx.x].theta) * sin(M_s[threadIdx.z][threadIdx.y][threadIdx.x].theta);

		//the field from random thermal motion
		H_t.x += Htherm_d[i].x;
		H_t.y += Htherm_d[i].y;
		H_t.z += Htherm_d[i].z;

		//the exchange field
		SphVector up, down, left, right, front, back;

		//if(i % (WIDTH * HEIGHT) < WIDTH) //if at top of particle
		if(ty == 0)
			up = M[i + WIDTH * (HEIGHT - 1)]; 
		else if(threadIdx.y > 0)
			up = M_s[threadIdx.z][threadIdx.y - 1][threadIdx.x];
		else
			up = M[i - WIDTH];

		//if(i % (WIDTH * HEIGHT) > (WIDTH * (HEIGHT - 1) - 1)) //if at bottom of particle
		if(ty == (HEIGHT - 1))
			down = M[i - WIDTH * (HEIGHT - 1)];
		else if(threadIdx.y < (blockDim.y - 1))
			down = M_s[threadIdx.z][threadIdx.y + 1][threadIdx.x];
		else
			down = M[i + WIDTH];	

		//if(i % WIDTH == 0) //if at left
		if(tx == 0)
			left = M[i + (WIDTH - 1)]; 
		else if(threadIdx.x > 0)
			left = M_s[threadIdx.z][threadIdx.y][threadIdx.x - 1];
		else
			left = M[i - 1];

		//if((i + 1) % WIDTH == 0) //if at right
		if(tx == (WIDTH - 1))
			right = M[i - (WIDTH - 1)];
		else if(threadIdx.x < (blockDim.x - 1))
			right = M_s[threadIdx.z][threadIdx.y][threadIdx.x + 1];
		else
			right = M[i + 1];

		//if(i < (WIDTH * HEIGHT)) //if at front
		if(tz == 0)
			front = M[i + (WIDTH * HEIGHT * (DEPTH - 1))];
		else if(threadIdx.z > 0)
			front = M_s[threadIdx.z - 1][threadIdx.y][threadIdx.x];
		else
			front = M[i - (WIDTH * HEIGHT)];

		//if(i >= (WIDTH * HEIGHT * (DEPTH - 1))) //if at rear
		if(tz == (DEPTH - 1))
			back = M[i - (WIDTH * HEIGHT * (DEPTH - 1))];
		else if(threadIdx.z < (blockDim.z - 1))
			back = M_s[threadIdx.z + 1][threadIdx.y][threadIdx.x];
		else
			back = M[i + (WIDTH * HEIGHT)];

		double Hex = 2.0 * JEX / (MSAT * ALEN * ALEN);

		H_t.x += Hex * (sin(up.theta) * cos(up.phi) + sin(down.theta) * cos(down.phi) + sin(left.theta) * cos(left.phi) + sin(right.theta) * cos(right.phi) + sin(front.theta) * cos(front.phi) + sin(back.theta) * cos(back.phi));
		H_t.y += Hex * (sin(up.theta) * sin(up.phi) + sin(down.theta) * sin(down.phi) + sin(left.theta) * sin(left.phi) + sin(right.theta) * sin(right.phi) + sin(front.theta) * sin(front.phi) + sin(back.theta) * sin(back.phi)); 
		H_t.z += Hex * (cos(up.theta) + cos(down.theta) + cos(left.theta) + cos(right.theta) + cos(front.theta) + cos(back.theta));

		//__syncthreads();
		H_d[i] = H_t;
	}
}

//Shamelessly copied from Numerical Recipes
/*
Given values for the variables y[1..n] and their derivatives dydx[1..n] known at x , use the
fourth-order Runge-Kutta method to advance the solution over an interval h and return the
incremented variables as yout[1..n] , which need not be a distinct array from y . The user
supplies the routine derivs(x,y,dydx) , which returns derivatives dydx at x .
*/
void rk4(SphVector y_d[], SphVector dydx_d[], int n, double x, double h, SphVector yout[], void (*derivs)(double, SphVector[], SphVector[], int, Vector[]), bool CopyToHost) {
	double xh, hh, h6; 
	dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
        dim3 gridDim(ceil(WIDTH/BLOCK_SIZE), ceil(HEIGHT/BLOCK_SIZE), ceil(DEPTH/BLOCK_SIZE));	

	//device arrays
	SphVector *dym_d, *dyt_d, *yt_d;

	//allocate device arrays
	hipMalloc((void **)&dym_d, sizeof(SphVector) * n);
	hipMalloc((void **)&dyt_d, sizeof(SphVector) * n);
	hipMalloc((void **)&yt_d, sizeof(SphVector) * n);

	hh = h * 0.5;
	h6 = h / 6.0;
	xh = x + hh;

	//First step
	rk4First<<<ceil(n/512.0), 512>>>(yt_d, y_d, dydx_d, hh, n);

	//Second step
	computeField<<<gridDim, blockDim>>>(H_d, H, Htherm_d, yt_d, n); 
	(*derivs)<<<ceil(n/512.0), 512>>>(xh, yt_d, dyt_d, n, H_d);
	rk4Second<<<ceil(n/512.0), 512>>>(yt_d, y_d, dyt_d, hh, n);

	//Third step
	computeField<<<gridDim, blockDim>>>(H_d, H, Htherm_d, yt_d, n); 
	(*derivs)<<<ceil(n/512.0), 512>>>(xh, yt_d, dym_d, n, H_d);
	rk4Third<<<ceil(n/512.0), 512>>>(yt_d, y_d, dym_d, dyt_d, h, n);

	//Fourth step
	computeField<<<gridDim, blockDim>>>(H_d, H, Htherm_d, yt_d, n); 
	(*derivs)<<<ceil(n/512.0), 512>>>(x + h, yt_d, dyt_d, n, H_d);
	//Accumulate increments with proper weights
	rk4Fourth<<<ceil(n/512.0), 512>>>(yout_d, y_d, dydx_d, dyt_d, dym_d, h6, n);

	//Copy yout to host
	if(CopyToHost)
		hipMemcpy(yout, yout_d, sizeof(SphVector) * n, hipMemcpyDeviceToHost);
	
	//Free device arrays
	hipFree(yt_d);
	hipFree(dyt_d);
	hipFree(dym_d);
	//hipFree(yout_d);
}

__global__ void computeHtherm(Vector * Htherm_d, int nvar, hiprandStateXORWOW_t * state) {
		int i = threadIdx.x + blockDim.x * blockIdx.x;

		if(i < nvar) {
			//the field from random thermal motion
			double vol = ALEN * ALEN * ALEN;
			double sd = (1e9) * sqrt((2 * BOLTZ * TEMP * ALPHA)/(GAMMA * vol * MSAT * TIMESTEP)); //time has units of s here

			double thermX = sd * hiprand_normal_double(&state[i]); 
			double thermY = sd * hiprand_normal_double(&state[i]);
			double thermZ = sd * hiprand_normal_double(&state[i]);

			Htherm_d[i].x += thermX;
			Htherm_d[i].y += thermY;
			Htherm_d[i].z += thermZ;
		}
}

__global__ void mDot(double t, SphVector M[], SphVector dMdt[], int nvar, Vector H[]) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	//Compute derivative
	if(i < nvar) {
		SphVector M_s = M[i];
		Vector H_s = H[i];

		dMdt[i].r = 0;
		dMdt[i].phi = GAMMA * ((cos(M_s.theta) * sin(M_s.phi) * H_s.y) / sin(M_s.theta) + (cos(M_s.theta) * cos(M_s.phi) * H_s.x) / sin(M_s.theta) - H_s.z) + ((ALPHA * GAMMA)/(1 + ALPHA * ALPHA)) * ((cos(M_s.phi) * H_s.y) / sin(M_s.theta) - (sin(M_s.phi) * H_s.x) / sin(M_s.theta));
		dMdt[i].theta = -GAMMA * (cos(M_s.phi) * H_s.y - sin(M_s.phi) * H_s.x) + ((ALPHA * GAMMA)/(1 + ALPHA * ALPHA)) * (cos(M_s.theta) * cos(M_s.phi) * H_s.x - H_s.z * sin(M_s.theta) + cos(M_s.theta) * sin(M_s.phi) * H_s.y);
	}
}

/*
Starting from initial values vstart[0..nvar-1] known at x1 use fourth-order Runge-Kutta
to advance nstep equal increments to x2. The user-supplied routine derivs(x,v,dvdx)
evaluates derivatives. Results are stored in the global variables y[0..nvar-1][0..nstep]
and xx[0..nstep].
*/
void rkdumb(SphVector vstart[], int nvar, double x1, double x2, int nstep, void (*derivs)(double, SphVector[], SphVector[], int, Vector[])) {
	double x, h;
	dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
        dim3 gridDim(ceil(WIDTH/BLOCK_SIZE), ceil(HEIGHT/BLOCK_SIZE), ceil(DEPTH/BLOCK_SIZE));	
	SphVector *v, *vout, *dv;

	//device arrays
	SphVector *v_d, *dv_d;

	v = (SphVector *)malloc(sizeof(SphVector) * nvar);
	vout = (SphVector *)malloc(sizeof(SphVector) * nvar);
	dv = (SphVector *)malloc(sizeof(SphVector) * nvar);

	hipMalloc((void **)&yout_d, sizeof(SphVector) * nvar);

	//allocate device memory for mDot
	hipMalloc((void **)&v_d, sizeof(SphVector) * nvar);
	hipMalloc((void **)&dv_d, sizeof(SphVector) * nvar);
	hipMalloc((void **)&H_d, sizeof(SphVector) * nvar);

	//allocate device memory for thermal motion
	hipMalloc((void **)&Htherm_d, sizeof(Vector) * nvar);

	for (int i = 0;i < nvar;i++) { 
		v[i] = vstart[i];
		y[i][0] = v[i]; 
	}

	xx[0] = x1;
	x = x1;
	h = (x2-x1)/nstep;

	for (int k = 0; k < nstep; k++) {

		//Copy memory to device
		//After the first timestep, the value of v and yout_d are the same. d2d memcpy is much faster than h2s, so do it instead
		if(k == 0) hipMemcpy(v_d, v, sizeof(SphVector) * nvar, hipMemcpyHostToDevice);
		//else hipMemcpy(v_d, yout_d, sizeof(SphVector) * nvar, hipMemcpyDeviceToDevice);
		else {
			SphVector *t_d = v_d;
			v_d = yout_d;
			yout_d = t_d;
		}

		//Generate thermal noise
		computeHtherm<<<ceil(nvar/512.0), 512>>>(Htherm_d, nvar, state);

		//Launch kernel to compute H field
		computeField<<<gridDim, blockDim>>>(H_d, H, Htherm_d, v_d, nvar); 

		//Launch kernel to compute derivatives
		(*derivs)<<<ceil(nvar/512.0), 512>>>(x, v_d, dv_d, nvar, H_d);
	
		bool CopyToHost = (k == (nstep - 1));

		//rk4(v_d,dv_d,nvar,x,h,vout,derivs);
		rk4(v_d,dv_d,nvar,x,h,vout,derivs, CopyToHost);
		if ((double)(x + h) == x) fprintf(stderr, "Step size too small in routine rkdumb");
		x += h;
		xx[k + 1] = x;
		for (int i = 0; i < nvar; i++) {
			v[i] = vout[i];
			y[i][k + 1] = v[i];
		}

	}

	free(dv);
	free(vout);
	free(v);
	hipFree(yout_d);
	hipFree(v_d);
	hipFree(dv_d);
	hipFree(H_d);
	hipFree(Htherm_d);
}

int main(int argc, char *argv[]) {
	int nvar = HEIGHT * WIDTH * DEPTH; //M for each particle 
	int nstep;
	double endTime;
	SphVector vstart[nvar]; 

	FILE * output = fopen("output.txt", "w");
	if(output == NULL) {
		printf("error opening file\n");
		return 0;
	}

	#if BENCHMARK
	FILE * times = fopen("times.txt", "w");
	if(times == NULL) {
		printf("error opening file: times.txt\n");
		return 1;
	}
	fprintf(times, "Time to simulate %fns\n");
	#endif

	//Initialize random number generator
	unsigned long long seed = time(NULL);
	hipMalloc((void **)&state, sizeof(hiprandStateXORWOW_t) * nvar);
	initializeRandom<<<ceil(nvar/512.0), 512>>>(state, nvar, seed);

	
	for(int i = 0; i < nvar; i++) {	
		vstart[i].r = MSAT;
		vstart[i].theta = 0.01;
		vstart[i].phi = 0;
	}

	Vector Happl = {0.0, 0.0, FIELDRANGE};
	endTime = FIELDTIMESTEP; 
	endTime /= 100; //Reduce host memory usage
	nstep = ((int)ceil(endTime/TIMESTEP));

	xx = (double *)malloc(sizeof(double) * (nstep + 1));
	y = (SphVector **)malloc(sizeof(SphVector *) * nvar); 
	for(int i = 0; i < nvar; i++) {
		y[i] = (SphVector *)malloc(sizeof(SphVector) * (nstep + 1));
	}

	
	bool isDecreasing = true;
	for(int i = 0; i <= (4 * (int)(FIELDRANGE/FIELDSTEP)); i++) {
		//Applied field
		H.x = Happl.x;
		H.y = Happl.y;
		H.z = Happl.z;

		#if BENCHMARK
		time_t start = time(NULL);
		#endif

		for(int j = 0; j < 100; j++) {
			//Simulate!
			rkdumb(vstart, nvar, endTime * j, endTime * (j + 1) - TIMESTEP, nstep, mDot); 

			for(int i = 0; i < nvar; i++) {
				vstart[i].r = y[i][nstep].r;
				vstart[i].theta = y[i][nstep].theta;
				vstart[i].phi = y[i][nstep].phi;
			}
		}
	
		#if BENCHMARK
		time_t end = time(NULL);
		fprintf(times, "%lds\n", (long)(end - start));
		fflush(times);
		#endif

		double mag = 0.0;	
		for(int k = 0; k < nvar; k++) {
			mag += (y[k][nstep].r)*cos(y[k][nstep].theta);
		}

		mag /= (double)nvar;
		fprintf(output, "%f\t%f\n", Happl.z, mag);
		fflush(output);

		//Adjust applied field strength at endTime intervals	
		if(Happl.z + FIELDRANGE < 1.0) isDecreasing = false;
		isDecreasing ? (Happl.z -= FIELDSTEP) : (Happl.z += FIELDSTEP);
	}
	//Probably don't really need these since we're about to exit the program
	free(xx);
	free(y);
	hipFree(state);
	return 0;
}
